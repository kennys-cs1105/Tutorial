#include "hip/hip_runtime.h"
#include <stdio.h>
#include "tools/common.cuh"

// CUDA事件计时

#define NUM_REPEATS 10

__device__ float add(const float x, const float y)
{
    return x + y;
}

__global__ void addFromGPU(float *A, float *B, float *C, const int N)
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int id = tid + bid * blockDim.x;

    if (id >= N) return;
    C[id] = add(A[id], B[id]);
}


void initialData(float *addr, int elemCount)
{
    for (int i = 0; i < elemCount; i++)
    {
        addr[i] = (float)(rand() & 0xFF) / 10.f;
    }
    return;
}


int main(void)
{
    // set gpu
    setGPU();

    // 分配主机内存 设备内存 并初始化
    int iElemCount = 4096;
    size_t stBytesCount = iElemCount * sizeof(float);

    float *fpHost_A, *fpHost_B, *fpHost_C;
    fpHost_A = (float *)malloc(stBytesCount);
    fpHost_B = (float *)malloc(stBytesCount);
    fpHost_C = (float *)malloc(stBytesCount);
    if (fpHost_A != NULL && fpHost_B != NULL && fpHost_C != NULL)
    {
        memset(fpHost_A, 0, stBytesCount);
        memset(fpHost_B, 0, stBytesCount);
        memset(fpHost_C, 0, stBytesCount);
    }
    else
    {
        printf("Fail to allocate host memory..\n");
        exit(-1);
    }

    float *fpDevice_A, *fpDevice_B, *fpDevice_C;
    ErrorCheck(hipMalloc((float **)&fpDevice_A, stBytesCount), __FILE__, __LINE__);
    ErrorCheck(hipMalloc((float **)&fpDevice_B, stBytesCount), __FILE__, __LINE__);
    ErrorCheck(hipMalloc((float **)&fpDevice_C, stBytesCount), __FILE__, __LINE__);
    if (fpDevice_A != NULL && fpDevice_B != NULL && fpDevice_C != NULL)
    {
        ErrorCheck(hipMemset(fpDevice_A, 0, stBytesCount), __FILE__, __LINE__);
        ErrorCheck(hipMemset(fpDevice_B, 0, stBytesCount), __FILE__, __LINE__);
        ErrorCheck(hipMemset(fpDevice_C, 0, stBytesCount), __FILE__, __LINE__);
    }
    else
    {
        printf("Fail to allocate memory..\n");
        free(fpHost_A);
        free(fpHost_B);
        free(fpHost_C);
        exit(-1);
    }

    // 初始化主机数据
    srand(666);
    initialData(fpHost_A, iElemCount);
    initialData(fpHost_B, iElemCount);

    // 数据从主机复制到设备
    ErrorCheck(hipMemcpy(fpDevice_A, fpHost_A, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__);
    ErrorCheck(hipMemcpy(fpDevice_B, fpHost_B, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__);
    ErrorCheck(hipMemcpy(fpDevice_C, fpHost_C, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__);

    // 核函数计算
    dim3 block(32);
    dim3 grid((iElemCount + block.x -1) / 32);

    float t_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        ErrorCheck(hipEventCreate(&start), __FILE__, __LINE__);
        ErrorCheck(hipEventCreate(&stop), __FILE__, __LINE__);
        ErrorCheck(hipEventRecord(start), __FILE__, __LINE__);
        hipEventQuery(start);

        addFromGPU<<<grid, block>>>(fpDevice_A, fpDevice_B, fpDevice_C, iElemCount);

        ErrorCheck(hipEventRecord(stop), __FILE__, __LINE__);
        ErrorCheck(hipEventSynchronize(stop), __FILE__, __LINE__);
        float elapsed_time;
        ErrorCheck(hipEventElapsedTime(&elapsed_time, start, stop), __FILE__, __LINE__);

         if (repeat > 0)
         {
            t_sum += elapsed_time;
         }

         ErrorCheck(hipEventDestroy(start), __FILE__, __LINE__);
         ErrorCheck(hipEventDestroy(stop), __FILE__, __LINE__);
    }

    const float t_ave = t_sum / NUM_REPEATS;
    printf("Time = %g ms.\n", t_ave);

    // 将计算得到的数据从设备传给主机
    ErrorCheck(hipMemcpy(fpHost_C, fpDevice_C, stBytesCount, hipMemcpyDeviceToHost), __FILE__, __LINE__);

    // 释放主机与设备内存
    free(fpHost_A);
    free(fpHost_B);
    free(fpHost_C);
    ErrorCheck(hipFree(fpDevice_A), __FILE__, __LINE__);
    ErrorCheck(hipFree(fpDevice_B), __FILE__, __LINE__);
    ErrorCheck(hipFree(fpDevice_C), __FILE__, __LINE__);

    ErrorCheck(hipDeviceReset(), __FILE__, __LINE__);
    return 0;
}